#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <string.h>
#include <png.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE  8

unsigned char *host_s = NULL;       // source image array
unsigned char *host_t = NULL;       // target image array

unsigned char *device_s = NULL;       //device source image array
unsigned char *device_t = NULL;       //device target image array
int *device_m = NULL;

FILE *fp_s = NULL;                  // source file handler
FILE *fp_t = NULL;                  // target file handler

unsigned int   width, height;       // image width, image height
unsigned int   rgb_raw_data_offset; // RGB raw data offset
unsigned char  bit_per_pixel;       // bit per pixel
unsigned short byte_per_pixel;      // byte per pixel

// bitmap header
unsigned char header[54] = {
	0x42,        // identity : B
	0x4d,        // identity : M
	0, 0, 0, 0,  // file size
	0, 0,        // reserved1
	0, 0,        // reserved2
	54, 0, 0, 0, // RGB data offset
	40, 0, 0, 0, // struct BITMAPINFOHEADER size
	0, 0, 0, 0,  // bmp width
	0, 0, 0, 0,  // bmp height
	1, 0,        // planes
	24, 0,       // bit per pixel
	0, 0, 0, 0,  // compression
	0, 0, 0, 0,  // data size
	0, 0, 0, 0,  // h resolution
	0, 0, 0, 0,  // v resolution 
	0, 0, 0, 0,  // used colors
	0, 0, 0, 0   // important colors
};

// sobel mask (5x5 version)
// Task 2: Put mask[][][] into Shared Memroy
int mask[MASK_N][MASK_X][MASK_Y] = {
	{{ -1, -4, -6, -4, -1},
	 { -2, -8,-12, -8, -2},
	 {  0,  0,  0,  0,  0},
	 {  2,  8, 12,  8,  2},
	 {  1,  4,  6,  4,  1}},
	{{ -1, -2,  0,  2,  1},
	 { -4, -8,  0,  8,  4},
	 { -6,-12,  0, 12,  6},
	 { -4, -8,  0,  8,  4},
	 { -1, -2,  0,  2,  1}}
};

int read_bmp (const char *fname_s) {
	fp_s = fopen(fname_s, "rb");
	if (fp_s == NULL) {
		printf("fopen fp_s error\n");
		return -1;
	}

	// move offset to 10 to find rgb raw data offset
	fseek(fp_s, 10, SEEK_SET);
	fread(&rgb_raw_data_offset, sizeof(unsigned int), 1, fp_s);

	// move offset to 18 to get width & height;
	fseek(fp_s, 18, SEEK_SET); 
	fread(&width,  sizeof(unsigned int), 1, fp_s);
	fread(&height, sizeof(unsigned int), 1, fp_s);

	// get bit per pixel
	fseek(fp_s, 28, SEEK_SET); 
	fread(&bit_per_pixel, sizeof(unsigned short), 1, fp_s);
	byte_per_pixel = bit_per_pixel / 8;

	// move offset to rgb_raw_data_offset to get RGB raw data
	fseek(fp_s, rgb_raw_data_offset, SEEK_SET);

	// Task 3: Assign host_s to Pinnned Memory
	// Hint  : err = cudaMallocHost ( ... )
	//         if (err != CUDA_SUCCESS)
	hipHostMalloc(&host_s, (size_t)width * height * byte_per_pixel, hipHostMallocDefault);
  /*host_s = (unsigned char *) malloc((size_t)width * height * byte_per_pixel);
	if (host_s == NULL) {
		printf("malloc images_s error\n");
		return -1;
	}*/
  
    
	// Task 3: Assign host_t to Pinned Memory
	// Hint  : err = cudaMallocHost ( ... )
	//         if (err != CUDA_SUCCESS)
	hipHostMalloc(&host_t, (size_t)width * height * byte_per_pixel, hipHostMallocDefault);
  /*host_t = (unsigned char *) malloc((size_t) width * height * byte_per_pixel);
	if (host_t == NULL) {
		printf("malloc host_t error\n");
		return -1;
	}*/
  
	fread(host_s, sizeof(unsigned char), (size_t)(long) width * height * byte_per_pixel, fp_s);

	return 0;
}

//void sobel () {
void __global__ sobel(const int width, const int height, const int byte_per_pixel, unsigned char* devices, unsigned char* devicet, int *mask) {
	int  x, y, i, v, u;            // for loop counter
	int  R, G, B;                  // color of R, G, B
	double val[MASK_N*3] = {0.0};
	int adjustX, adjustY, xBound, yBound;

	// Task 2: Put mask[][][] into Shared Memory
	// Hint  : Please declare it in kernel function
	//         Then use some threads to move data from global memory to shared memory
	//         Remember to __syncthreads() after it's done <WHY?>
  extern __shared__ int mask_shared[];
  if(threadIdx.x<50)
		mask_shared[threadIdx.x] = mask[threadIdx.x];
	// Task 1: Relabel x or y or both into combination of blockIdx, threadIdx ... etc
	// Hint A: We do not have enough threads for each pixels in the image, so what should we do?
	// Hint B: Maybe you can map each y to different threads in different blocks
	y = blockIdx.x * blockDim.x + threadIdx.x;
  if (y >= height) 
        return; 
  //for (y = 0; y < height; ++y) 
  {  
		for (x = 0; x < width; ++x) {
			for (i = 0; i < MASK_N; ++i) {
				adjustX = (MASK_X % 2) ? 1 : 0;
				adjustY = (MASK_Y % 2) ? 1 : 0;
				xBound = MASK_X /2;
				yBound = MASK_Y /2;

				val[i*3+2] = 0.0;
				val[i*3+1] = 0.0;
				val[i*3] = 0.0;

				for (v = -yBound; v < yBound + adjustY; ++v) {
					for (u = -xBound; u < xBound + adjustX; ++u) {
						if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
							R = devices[byte_per_pixel * (width * (y+v) + (x+u)) + 2];
							G = devices[byte_per_pixel * (width * (y+v) + (x+u)) + 1];
							B = devices[byte_per_pixel * (width * (y+v) + (x+u)) + 0];
							val[i*3+2] += R * mask_shared[i* MASK_Y * MASK_X + (u + xBound) * MASK_Y + (v + yBound)];//val[i*3+2] += R * mask_shared[i][u + xBound][v + yBound];
							val[i*3+1] += G * mask_shared[i* MASK_Y * MASK_X + (u + xBound) * MASK_Y + (v + yBound)];//val[i*3+1] += G * mask_shared[i][u + xBound][v + yBound];
							val[i*3+0] += B * mask_shared[i* MASK_Y * MASK_X + (u + xBound) * MASK_Y + (v + yBound)];//val[i*3+0] += B * mask_shared[i][u + xBound][v + yBound];
						}	
					}
				}
			}

			double totalR = 0.0;
			double totalG = 0.0;
			double totalB = 0.0;
			for (i = 0; i < MASK_N; ++i) {
				totalR += val[i * 3 + 2] * val[i * 3 + 2];
				totalG += val[i * 3 + 1] * val[i * 3 + 1];
				totalB += val[i * 3 + 0] * val[i * 3 + 0];
			}

			totalR = sqrt(totalR) / SCALE;
			totalG = sqrt(totalG) / SCALE;
			totalB = sqrt(totalB) / SCALE;
			const unsigned char cR = (totalR > 255.0) ? 255 : totalR;
			const unsigned char cG = (totalG > 255.0) ? 255 : totalG;
			const unsigned char cB = (totalB > 255.0) ? 255 : totalB;
			devicet[byte_per_pixel * (width * y + x) + 2] = cR;
			devicet[byte_per_pixel * (width * y + x) + 1] = cG;
			devicet[byte_per_pixel * (width * y + x) + 0] = cB;
		}
	}
}

int write_bmp (const char *fname_t) {
	unsigned int file_size;

	fp_t = fopen(fname_t, "wb");
	if (fp_t == NULL) {
		printf("fopen fname_t error\n");
		return -1;
	}

	// file size  
	file_size = width * height * byte_per_pixel + rgb_raw_data_offset;
	header[2] = (unsigned char)(file_size & 0x000000ff);
	header[3] = (file_size >> 8)  & 0x000000ff;
	header[4] = (file_size >> 16) & 0x000000ff;
	header[5] = (file_size >> 24) & 0x000000ff;

	// width
	header[18] = width & 0x000000ff;
	header[19] = (width >> 8)  & 0x000000ff;
	header[20] = (width >> 16) & 0x000000ff;
	header[21] = (width >> 24) & 0x000000ff;

	// height
	header[22] = height &0x000000ff;
	header[23] = (height >> 8)  & 0x000000ff;
	header[24] = (height >> 16) & 0x000000ff;
	header[25] = (height >> 24) & 0x000000ff;

	// bit per pixel
	header[28] = bit_per_pixel;

	// write header
	fwrite(header, sizeof(unsigned char), rgb_raw_data_offset, fp_t);

	// write image
	fwrite(host_t, sizeof(unsigned char), (size_t)(long)width * height * byte_per_pixel, fp_t);

	fclose(fp_s);
	fclose(fp_t);

	return 0;
}

int main(int argc, char **argv) {

    assert(argc == 3);
    const char* input = argv[1];
    const char* output = argv[2];
	  read_bmp(input); // 24 bit gray level image
    
    
    hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);
		int num_thread = prop.maxThreadsPerBlock; 
	// Task 1: Allocate memory on GPU
	// Hint  : cudaMalloc ()
	//         What do we need to store on GPU? (input image, output image, ...)
    hipMalloc(&device_t, (size_t)width * height * byte_per_pixel);
		hipMalloc(&device_s, (size_t)width * height * byte_per_pixel);
	// Task 1: Memory copy from Host to Device (GPU)
	// Hint  : cudaMemcpy ( ... , cudaMemcpyHostToDevice )
    hipMemcpy(device_s, host_s, (size_t)width * height * byte_per_pixel, hipMemcpyHostToDevice);
	// Task 1: Modify sobel() to CUDA kernel function
	// Hint  : sobel_Kernel <<< ??? , ??? >>> ( ??? );
	  hipMalloc(&device_m, (size_t)MASK_N * MASK_X * MASK_Y * sizeof(int));
		hipMemcpy(device_m, mask, (size_t)MASK_N * MASK_X * MASK_Y * sizeof(int), hipMemcpyHostToDevice);
    sobel<<<height, num_thread, 50*sizeof(int)>>>(width, height, byte_per_pixel, device_s, device_t, device_m);//sobel();

	// Task 1: Memory Copy from Device (GPU) to Host
	// Hint  : cudaMemcpy ( ... , cudaMemcpyDeviceToHost )
    hipMemcpy(host_t, device_t, (size_t)width * height * byte_per_pixel, hipMemcpyDeviceToHost);
		
	// Task 1: Free memory on device
	// Hint  : cudaFree ( ... )    
		hipFree(device_t);
		hipFree(device_s);
    hipFree(device_m);
	  write_bmp(output);

	// Task 3: Free Pinned memory
	// Hint  : replace free ( ... ) by cudaFreeHost ( ... )
	  hipHostFree(host_s);//free (host_s);
	  hipHostFree(host_t);//free (host_t);
}
